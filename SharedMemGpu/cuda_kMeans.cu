#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_kMeans.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "timer.hpp"

/**
 * Kernel:      initializeSums_kernel
 * 
 * In Args:     int centroids_size, int* nPoints, double* sumX, double* sumY, double* sumZ
 * Desc:        ONLY RUNS ON 1 THREAD, resets device values for nPoints, sumX, Y, and Z
*/
__global__ 
void initializeSums_kernel(int centroids_size, int* nPoints, double* sumX, double* sumY, double* sumZ)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx == 0)
    {
        // reset with zeroes
        for (int j = 0; j < centroids_size; j++) {
            nPoints[j] = 0;
            sumX[j] = 0;
            sumY[j] = 0;
            sumZ[j] = 0;
        }
    }
}

/**
 * Kernel:      kMeansClustering_kernel
 * 
 * In Args:     Point* points, Point* centroids, int points_size, int centroids_size, int* nPoints,
                double* sumX, double* sumY, double* sumZ
 * Desc:        Carries out the KMeansClustering on points, assigning a cluster to each point
*/
__global__
void kMeansClustering_kernel(Point* points, Point* centroids, int points_size, int centroids_size, int* nPoints,
                            double* sumX, double* sumY, double* sumZ)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Declare shared memory arrays for each block
    __shared__ int shared_nPoints[3];
    __shared__ double shared_sumX[3];
    __shared__ double shared_sumY[3];
    __shared__ double shared_sumZ[3];

    // Initialize shared memory for each cluster
    if (threadIdx.x < centroids_size) {
        shared_nPoints[threadIdx.x] = 0;
        shared_sumX[threadIdx.x] = 0.0;
        shared_sumY[threadIdx.x] = 0.0;
        shared_sumZ[threadIdx.x] = 0.0;
    }

    __syncthreads();

    if (idx < points_size) {
        for (int clusterId = 0; clusterId < centroids_size; clusterId++) {
            // quick hack to get cluster index
            Point c = centroids[clusterId];
            Point p = points[idx];
            // Distance() funtion brought in due to being __host__
            double dist = (p.x - c.x) * (p.x - c.x) + (p.y - c.y) * (p.y - c.y) +
                    (p.z - c.z) * (p.z - c.z);
            if (dist < p.minDist) {
                p.minDist = dist;
                p.cluster = clusterId;
            }
            points[idx] = p;
        }

        // Append data to shared memory in the block
        int clusterId = points[idx].cluster;
        atomicAdd(&shared_nPoints[clusterId], 1);
        atomicAdd(&shared_sumX[clusterId], points[idx].x);
        atomicAdd(&shared_sumY[clusterId], points[idx].y);
        atomicAdd(&shared_sumZ[clusterId], points[idx].z);

        points[idx].minDist = __DBL_MAX__; // reset distance
    }

    __syncthreads(); // Synchronize threads within the block before updating global memory

    // Update global memory with results from shared memory
    if (threadIdx.x < centroids_size) {
        atomicAdd(&nPoints[threadIdx.x], shared_nPoints[threadIdx.x]);
        atomicAdd(&sumX[threadIdx.x], shared_sumX[threadIdx.x]);
        atomicAdd(&sumY[threadIdx.x], shared_sumY[threadIdx.x]);
        atomicAdd(&sumZ[threadIdx.x], shared_sumZ[threadIdx.x]);
    }
}

/**
 * Kernel:      ResetClusters_kernel
 * 
 * In Args:     Point* centroids, int centroids_size, int* nPoints, double* sumX, double* sumY, double* sumZ
 * Desc:        ONLY RUNS 1 thread, selects new centroid points on device
*/
__global__
void ResetClusters_kernel(Point* centroids, int centroids_size, int* nPoints, double* sumX, double* sumY, double* sumZ)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx == 0)
    {
        // Compute the new centroids
        for (int clusterId = 0; clusterId < centroids_size; clusterId++) {
            centroids[clusterId].x = sumX[clusterId] / nPoints[clusterId];
            centroids[clusterId].y = sumY[clusterId] / nPoints[clusterId];
            centroids[clusterId].z = sumZ[clusterId] / nPoints[clusterId];
    }
    }
}

/**
 * wrapper:     Cuda_KMeans::do_cuda_kMeans
 * 
 * In Args:     int epochs, int k, char* category1,
                char* category2, char* category3, 
                Point* points, Point* centroids, int points_size, int centroids_size
 * Desc:        Setup for device, allocates cuda memory for GPU implementation of KMeansClustering
*/
namespace Cuda_KMeans {
	void do_cuda_kMeans(int epochs, int k, char* category1,
                      char* category2, char* category3, 
                      Point* points, Point* centroids, int points_size, int centroids_size)
	{

        // Allocate cuda memory for points and centroids
        Point* cuda_points, *cuda_centroids;
        hipMalloc((void**)&cuda_points, points_size * sizeof(Point));
        hipMalloc((void**)&cuda_centroids, centroids_size * sizeof(Point));
        hipMemcpy(cuda_points, points, points_size * sizeof(Point), hipMemcpyHostToDevice);

        int* cuda_nPoints;
        double* cuda_sumX, *cuda_sumY, *cuda_sumZ;
        hipMalloc((void**)&cuda_nPoints, k * sizeof(int));
        hipMalloc((void**)&cuda_sumX, k * sizeof(double));
        hipMalloc((void**)&cuda_sumY, k * sizeof(double));
        hipMalloc((void**)&cuda_sumZ, k * sizeof(double));

        int* nPoints = new int[k];
        double* sumX = new double[k];
        double* sumY = new double[k];
        double* sumZ = new double[k];

        // Initialize with zeroes
        for (int j = 0; j < k; j++) {
            nPoints[j] = 0;
            sumX[j] = 0;
            sumY[j] = 0;
            sumZ[j] = 0;
        }

        hipMemcpy(cuda_centroids, centroids, centroids_size * sizeof(Point), hipMemcpyHostToDevice);
        hipMemcpy(cuda_nPoints, nPoints, k * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(cuda_sumX, sumX, k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(cuda_sumY, sumY, k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(cuda_sumZ, sumZ, k * sizeof(double), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (points_size + blockSize - 1) / blockSize;

	  double startTime = get_wall_time();
        for (int e = 0; e < epochs; e++)
        {
            printf("Cuda Epoch %d\n", e);
            // Launch Kernels
            initializeSums_kernel<<<1,1>>>(centroids_size, cuda_nPoints, cuda_sumX, cuda_sumY, cuda_sumZ);
            hipDeviceSynchronize();
            kMeansClustering_kernel<<<numBlocks, blockSize>>>(cuda_points, cuda_centroids, points_size, centroids_size, cuda_nPoints, cuda_sumX, cuda_sumY, cuda_sumZ);
            hipDeviceSynchronize();
            ResetClusters_kernel<<<1,1>>>(cuda_centroids, centroids_size, cuda_nPoints, cuda_sumX, cuda_sumY, cuda_sumZ);
            hipDeviceSynchronize();
        }
	  double endTime = get_wall_time();
	  double totalTime = endTime - startTime;
	  double averageTime = totalTime / epochs;
	  printf("Algorithm took %f seconds to complete and averaged %f seconds per epoch\n", totalTime, averageTime);

        // Copy points memory from device to host
        hipMemcpy(points, cuda_points, points_size * sizeof(Point), hipMemcpyDeviceToHost);
        hipMemcpy(centroids, cuda_centroids, points_size * sizeof(Point), hipMemcpyDeviceToHost);
        hipFree(cuda_points);
        hipFree(cuda_centroids);
        hipFree(cuda_nPoints);
        hipFree(cuda_sumX);
        hipFree(cuda_sumY);
        hipFree(cuda_sumZ); 
	}
}
